/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  float x = (float)i;
  float s = sinf(x); 
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s*s+c*c);
}

int main(int argc, char **argv)
{
  const long blockSize = 1024, msize = 128;
  const long n = 1024 * blockSize * msize;
  const long bytes = n * sizeof(float);

  printf("Mem used for transfer: %luMB\n", bytes/1024/1024);
   
  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );
  
  // allocate pinned host memory and device memory
  float *a, *d_a, *h_a;
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) );      // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device
  h_a = (float *)malloc(sizeof(float) * bytes);
  if (!h_a)
  {
	  printf("allocate pageable host memory failed");
	  return -1;
  }

  float ms; // elapsed time in milliseconds
  
  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  checkCuda( hipStreamCreate(&stream) );
  
  // baseline case - sequential transfer and execute pinned
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) );
  kernel<<<n/blockSize, blockSize>>>(d_a, 0);
  checkCuda( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for sequential transfer pinned and execute (ms): %f\n", ms);

  // baseline case - sequential transfer and execute pageable
  memset(h_a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice) );
  kernel<<<n/blockSize, blockSize>>>(d_a, 0);
  checkCuda( hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for sequential transfer pageable and execute (ms): %f\n", ms);


  // asynchronous version  {copy, kernel, copy} pinned
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipMemcpyAsync(&d_a, &a, 
			  bytes, hipMemcpyHostToDevice, 
			  stream) );
  kernel<<<n/blockSize, blockSize, 0, stream>>>(d_a, 0);
  checkCuda( hipMemcpyAsync(&a, &d_a, 
			  bytes, hipMemcpyDeviceToHost,
			  stream) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous transfer pinned and execute (ms): %f\n", ms);

  // asynchronous version  {copy, kernel, copy} pageable
  memset(h_a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipMemcpyAsync(&d_a, &h_a, 
			  bytes, hipMemcpyHostToDevice, 
			  stream) );
  kernel<<<n/blockSize, blockSize, 0, stream>>>(d_a, 0);
  checkCuda( hipMemcpyAsync(&h_a, &d_a, 
			  bytes, hipMemcpyDeviceToHost,
			  stream) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous transfer pageable and execute (ms): %f\n", ms);

  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  checkCuda( hipStreamDestroy(stream) );
  hipFree(d_a);
  hipHostFree(a);
  free(h_a);

  return 0;
}
